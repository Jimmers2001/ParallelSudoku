#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <algorithm>
#include <cctype>
#include <string>
#include <numeric>
#include <cmath>
 
/*
__global__ void __multiply__ (){
    printf("In multiply\n");
}*/


     
extern "C++"  void CudaThings(char *sudoku_str){
  int i, j, k, l, num;
  int sudoku_matrix[9][9];
  char temp[2];
  char *ptr;
  int updated = 0;

  // Convert string representation to matrix representation
  ptr = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          strncpy(temp, ptr++, 1);
          sudoku_matrix[i][j] = atoi(temp);
      }
  }
  
  // Perform elimination
  char *value = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          if (sudoku_matrix[i][j] != 0) {
              continue;
          }
              
          int possible_values[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
          int num_possible_values = 9;
          
          // Check row
          for (k = 0; k < 9; k++) {
              num = sudoku_matrix[i][k];
              if (num != 0) {
                  for (l = 0; l < num_possible_values; l++) {
                      if (possible_values[l] == num) {
                          possible_values[l] = possible_values[--num_possible_values];
                          break;
                      }
                  }
              }
          }
          
          // Check column
          for (k = 0; k < 9; k++) {
              num = sudoku_matrix[k][j];
              if (num != 0) {
                  for (l = 0; l < num_possible_values; l++) {
                      if (possible_values[l] == num) {
                          possible_values[l] = possible_values[--num_possible_values];
                          break;
                      }
                  }
              }
          }
          
          // Check 3x3 square
          int square_i = (i / 3) * 3;
          int square_j = (j / 3) * 3;
          for (k = square_i; k < square_i + 3; k++) {
              for (l = square_j; l < square_j + 3; l++) {
                  num = sudoku_matrix[k][l];
                  if (num != 0) {
                      for (int m = 0; m < num_possible_values; m++) {
                          if (possible_values[m] == num) {
                              possible_values[m] = possible_values[--num_possible_values];
                              break;
                          }
                      }
                  }
              }
          }
          
          // If only one possible value, fill cell
          if (num_possible_values == 1) {
              sudoku_matrix[i][j] = possible_values[0];
              updated = 1;
              break;
          }
      }
      if (updated) {
          break;
      }
  }
  value++;
  
  // Convert matrix representation back to string representation
  ptr = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          sprintf(temp, "%d", sudoku_matrix[i][j]);
          strncpy(ptr++, temp, 1);
      }
  }
  
  value = sudoku_str;
}