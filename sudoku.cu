#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <algorithm>
#include <cctype>
#include <string>
#include <numeric>
#include <cmath>
     
extern "C++" void CudaElimination(char *sudoku_str){
  int i, j, k, l, num;
  int sudoku_matrix[9][9];
  char temp[2];
  char *ptr;
  int updated = 0;

  ///////////////////////////////////////////////////////////////
  //                    Making String a Matrix                //
  //////////////////////////////////////////////////////////////
  ptr = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          strncpy(temp, ptr++, 1);
          sudoku_matrix[i][j] = atoi(temp);
      }
  }
  
  ///////////////////////////////////////////////////////////////
  //                           ELIM BELOW                     //
  //////////////////////////////////////////////////////////////
  char *value = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          if (sudoku_matrix[i][j] != 0) {
              continue;
          }
              
          int possible_numbers[9] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
          int num_possible_numbers = 9;
          
          
          for (k = 0; k < 9; k++) {
              num = sudoku_matrix[i][k];
              if (num != 0) {
                  for (l = 0; l < num_possible_numbers; l++) {
                      if (possible_numbers[l] == num) {
                          possible_numbers[l] = possible_numbers[--num_possible_numbers];
                          break;
                      }
                  }
              }
          }
          

          for (k = 0; k < 9; k++) {
              num = sudoku_matrix[k][j];
              if (num != 0) {
                  for (l = 0; l < num_possible_numbers; l++) {
                      if (possible_numbers[l] == num) {
                          possible_numbers[l] = possible_numbers[--num_possible_numbers];
                          break;
                      }
                  }
              }
          }
          
          // Check 3x3 block
          int block_i = (i / 3) * 3;
          int block_j = (j / 3) * 3;
          for (k = block_i; k < block_i + 3; k++) {
              for (l = block_j; l < block_j + 3; l++) {
                  num = sudoku_matrix[k][l];
                  if (num != 0) {
                      for (int m = 0; m < num_possible_numbers; m++) {
                          if (possible_numbers[m] == num) {
                              possible_numbers[m] = possible_numbers[--num_possible_numbers];
                              break;
                          }
                      }
                  }
              }
          }
          
          // If only one possible value, fill cell
          if (num_possible_numbers == 1) {
              sudoku_matrix[i][j] = possible_numbers[0];
              updated = 1;
              break;
          }
      }
      if (updated) {
          break;
      }
  }
  value++;
 /////////////////////////////////////////////////// 
  //  CONVERT BACK TO STRING TO SEND TO MAIN //
///////////////////////////////////////////////////
  ptr = sudoku_str;
  for (i = 0; i < 9; i++) {
      for (j = 0; j < 9; j++) {
          sprintf(temp, "%d", sudoku_matrix[i][j]);
          strncpy(ptr++, temp, 1);
      }
  }
  
  //set to global shared mem
  value = sudoku_str;

}
/*
int main(){
    return 0;
}*/